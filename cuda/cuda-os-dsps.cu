#include "hip/hip_runtime.h"
// dsps
#include "cuda-twoint-core-os.h"

__device__ void gpu_hrr_clear_os_dsps( double *eh ) {
    int i;
    // (DS|PS)
#pragma unroll
    for ( i=0; i<(0+18); i++ ) eh[i] = 0.e0;
}

__device__ void gpu_hrr_coef_os_dsps(
        const double *eh, double *DINT ) {
    int i, j, k, l, iao, jao, kao, lao, ix, iy;
    double coef_a, coef_ab, coef_abc;
    ix = 0;
#pragma unroll
    for ( i=0, iao=4; i<6; i++, iao++ ) {
        coef_a = LDG(DFACT[iao]);
        coef_ab = coef_a;
#pragma unroll
        for ( k=0, kao=1; k<3; k++, kao++ ) {
            coef_abc = coef_ab * LDG(DFACT[kao]);
            //DINT[ix] = coef_abc * eh[ix];
            //ix++;
            DINT[ix++] *= coef_abc;
        }
    }
}

__device__ void gpu_vrr_calc_os_dsps(
        const double T, const double cssss,
        const double zeta2, const double eta2, const double ze2,
        const double rz, const double re,
        const double PA[3], const double WP[3],
        const double QC[3], const double WQ[3],
        double *eh ) {
    double tmp0,tmp1;
    double ev[12];
    double ep[9];
    // (ss|ss) m=0,3
    //gpu_fmt( &ev[0], 3, T, cssss );
#if   CUDA_FMT_M == 3
    gpu_fmt3_method3( T, cssss, ev );
#elif CUDA_FMT_M == 2
    gpu_fmt3_method2( T, cssss, ev );
#elif CUDA_FMT_M == 1
    gpu_fmt3_method1( T, cssss, ev );
#else
    gpu_fmt3( ev, T, cssss );
#endif
    tmp0 = zeta2*(ev[0]-rz*ev[1]);
    tmp1 = zeta2*(ev[1]-rz*ev[2]);
    // (ps|ss) m=0,2
    ep[ 0]=PA[0]*ev[0]+WP[0]*ev[1];
    ep[ 1]=PA[1]*ev[0]+WP[1]*ev[1];
    ep[ 2]=PA[2]*ev[0]+WP[2]*ev[1];
    ep[ 3]=PA[0]*ev[1]+WP[0]*ev[2];
    ep[ 4]=PA[1]*ev[1]+WP[1]*ev[2];
    ep[ 5]=PA[2]*ev[1]+WP[2]*ev[2];
    ep[ 6]=PA[0]*ev[2]+WP[0]*ev[3];
    ep[ 7]=PA[1]*ev[2]+WP[1]*ev[3];
    ep[ 8]=PA[2]*ev[2]+WP[2]*ev[3];
    // (ds|ss) m=0,1
    ev[ 0]=PA[0]*ep[0]+WP[0]*ep[ 3]+tmp0;
    ev[ 1]=PA[0]*ep[3]+WP[0]*ep[ 6]+tmp1;
    ev[ 2]=PA[1]*ep[1]+WP[1]*ep[ 4]+tmp0;
    ev[ 3]=PA[1]*ep[4]+WP[1]*ep[ 7]+tmp1;
    ev[ 4]=PA[2]*ep[2]+WP[2]*ep[ 5]+tmp0;
    ev[ 5]=PA[2]*ep[5]+WP[2]*ep[ 8]+tmp1;
    ev[ 6]=PA[0]*ep[1]+WP[0]*ep[ 4];
    ev[ 7]=PA[0]*ep[4]+WP[0]*ep[ 7];
    ev[ 8]=PA[0]*ep[2]+WP[0]*ep[ 5];
    ev[ 9]=PA[0]*ep[5]+WP[0]*ep[ 8];
    ev[10]=PA[1]*ep[2]+WP[1]*ep[ 5];
    ev[11]=PA[1]*ep[5]+WP[1]*ep[ 8];
    // (ds|ps) m=[0,0]
    eh[ 0]+=QC[0]*ev[ 0]+WQ[0]*ev[ 1]+2.e0*ze2*ep[3];
    eh[ 1]+=QC[1]*ev[ 0]+WQ[1]*ev[ 1];
    eh[ 2]+=QC[2]*ev[ 0]+WQ[2]*ev[ 1];
    eh[ 3]+=QC[0]*ev[ 2]+WQ[0]*ev[ 3];
    eh[ 4]+=QC[1]*ev[ 2]+WQ[1]*ev[ 3]+2.e0*ze2*ep[4];
    eh[ 5]+=QC[2]*ev[ 2]+WQ[2]*ev[ 3];
    eh[ 6]+=QC[0]*ev[ 4]+WQ[0]*ev[ 5];
    eh[ 7]+=QC[1]*ev[ 4]+WQ[1]*ev[ 5];
    eh[ 8]+=QC[2]*ev[ 4]+WQ[2]*ev[ 5]+2.e0*ze2*ep[5];
    eh[ 9]+=QC[0]*ev[ 6]+WQ[0]*ev[ 7]+     ze2*ep[4];
    eh[10]+=QC[1]*ev[ 6]+WQ[1]*ev[ 7]+     ze2*ep[3];
    eh[11]+=QC[2]*ev[ 6]+WQ[2]*ev[ 7];
    eh[12]+=QC[0]*ev[ 8]+WQ[0]*ev[ 9]+     ze2*ep[5];
    eh[13]+=QC[1]*ev[ 8]+WQ[1]*ev[ 9];
    eh[14]+=QC[2]*ev[ 8]+WQ[2]*ev[ 9]+     ze2*ep[3];
    eh[15]+=QC[0]*ev[10]+WQ[0]*ev[11];
    eh[16]+=QC[1]*ev[10]+WQ[1]*ev[11]+     ze2*ep[5];
    eh[17]+=QC[2]*ev[10]+WQ[2]*ev[11]+     ze2*ep[4];
}

__device__ void gpu_vrr_cint_os_dsps( const double *ev, double *eh ) {
    int La=2, Lb=0, Lc=1, Ld=0;
    int i, ih, iv;
    // (DS|PS)
#pragma unroll
    //for ( i=0, iv=25, ih=0; i<18; i++, iv++, ih++ ) eh[ih]+=ev[iv];
    for ( i=0; i<18; i++) eh[i]+=ev[i];
}

__device__ void gpu_twoint_core_os_dsps(
//        const int *pLa, const int *pLb, const int *pLc, const int *pLd,
        const int *nijps, const double vzeta[], const double vdkab[],
        const double vxiza[], const double BA[3],
        const int *nklps, const double veta[], const double vdkcd[],
        const double vxizc[], const double DC[3], const double AC[3],
        double *DINT ) {
    int ijps, klps, i;
    double cssss, zeta, dkab, xiza, eta, xizc, dk, T;
    double zeta2, eta2, ze2, rz, re, PA[3], WP[3], QC[3], WQ[3];
    double PQ2, sqrho, rho, PC[3], QP[3];
//    double ev[18];
//    int La=*pLa, Lb=*pLb, Lc=*pLc, Ld=*pLd;

    gpu_hrr_clear_os_dsps( DINT );
    for ( ijps=0; ijps<(*nijps); ijps++ ) {
        zeta  = LDG(vzeta[ijps]);
        dkab  = LDG(vdkab[ijps]);
        xiza  = LDG(vxiza[ijps]);
        zeta2 = HALF * zeta;
#pragma unroll
        for ( i=0; i<3; i++ ) {
            PC[i] = AC[i] + xiza*BA[i];
            PA[i] = xiza * BA[i];
        }
        for ( klps=0; klps<(*nklps); klps++ ) {
            eta  = LDG(veta[klps]);
            dk   = dkab * LDG(vdkcd[klps]);
            xizc = LDG(vxizc[klps]);
            PQ2  = ZERO;
#pragma unroll
            for ( i=0; i<3; i++ ) {
                QC[i] = xizc*DC[i];
                QP[i] = xizc*DC[i] - PC[i];
                PQ2  += QP[i]*QP[i];
            }
            sqrho = sqrt(1.e0/(zeta+eta));
            rho   = sqrho*sqrho;
            rz    = rho * zeta;
            ze2 = rz * eta * HALF;
#pragma unroll
            for ( i=0; i<3; i++ ) {
                WP[i] = rz*QP[i];
                WQ[i] = rz*QP[i] - QP[i];
            }
            T     = rho * PQ2;
            cssss = sqrho * dk;
            gpu_vrr_calc_os_dsps(
                    T, cssss, zeta2, eta2, ze2, rz, re, PA, WP, QC, WQ,
                    //ev );
                    DINT );
            //gpu_vrr_cint_os_dsps( ev, DINT );
        }	// for (klps)
    }	// for (ijps)
    gpu_hrr_coef_os_dsps( DINT, DINT );
}

#if 0
int gpu_twoint_os_dsps(
        const int *pnworkers, const int *pworkerid,
        const int *pLa, const int *pLb, const int *pLc, const int *pLd,
        const int shel_atm[], const int shel_ini[],
        const double atom_x[], const double atom_y[],
        const double atom_z[], const int leading_cs_pair[],
        const double csp_schwarz[],
        const int csp_ics[], const int csp_jcs[],
        const int csp_leading_ps_pair[],
        const double psp_zeta[], const double psp_dkps[],
        const double psp_xiza[],
        // for partially direct SCF
        const long *pebuf_max_nzeri, long *ebuf_non_zero_eri,
        double ebuf_val[], short int ebuf_ind4[],
        int *last_ijcs, int *last_klcs ) {
    int Lab, Lcd, i, j, k, l, ipat, ix;
    int I2, IJ, K2, KL;
    int ijcs, ijcs0, ijcs1;
    int klcs, klcs0, klcs1, max_klcs;
    int ijps0, nijps, klps0, nklps;
    int ics, iat, iao, iao0, jcs, jat, jao, jao0;
    int kcs, kat, kao, kao0, lcs, lat, lao, lao0;
    double A[3], B[3], C[3], D[3], BA[3], DC[3], AC[3];
    double val_ab, val_cd, coe, coe0;
    double DINTEG[6*1*3*1];
    long nzeri, max_nzeri, nzeri4;
    int nworkers=*pnworkers, workerid=*pworkerid;
    int La=*pLa, Lb=*pLb, Lc=*pLc, Ld=*pLd;
    long ebuf_max_nzeri = *pebuf_max_nzeri;
    int mythread;

    
    mythread = omp_get_thread_num();
    if ( DFACT == NULL ) DFACT = gpu_getadd_dfact();
    
    Lab = La*(La+1)/2+Lb;
    Lcd = Lc*(Lc+1)/2+Ld;
    ijcs0 = leading_cs_pair[Lab];
    ijcs1 = leading_cs_pair[Lab+1];
    klcs0 = leading_cs_pair[Lcd];
    klcs1 = leading_cs_pair[Lcd+1];
    nzeri     = *ebuf_non_zero_eri;
    max_nzeri = ebuf_max_nzeri - 6*1*3*1;
    nzeri4    = nzeri*4;
    if ( nzeri >= max_nzeri ) {
        *last_ijcs = ijcs0+workerid;
        *last_klcs = klcs0 - 1;
        *ebuf_non_zero_eri = nzeri;
        return OFMO_EBUF_FULL;
    }

    for ( ijcs=ijcs0+workerid; ijcs<ijcs1; ijcs+=nworkers ) {
        val_ab = csp_schwarz[ijcs];
        ics    = csp_ics[ijcs];
        jcs    = csp_jcs[ijcs];
        ijps0  = csp_leading_ps_pair[ijcs];
        nijps  = csp_leading_ps_pair[ijcs+1]-ijps0;
        iat    = shel_atm[ics];
        jat    = shel_atm[jcs];
        iao0   = shel_ini[ics];
        jao0   = shel_ini[jcs];
        A[0]=atom_x[iat]; A[1]=atom_y[iat]; A[2]=atom_z[iat];
        B[0]=atom_x[jat]; B[1]=atom_y[jat]; B[2]=atom_z[jat];
        for ( i=0; i<3; i++ ) BA[i] = B[i] - A[i];
        max_klcs = ( Lab == Lcd ? ijcs+1 : klcs1 );
        for ( klcs=klcs0; klcs<max_klcs; klcs++ ) {
            val_cd = csp_schwarz[klcs];
            if ( val_ab*val_cd < EPS_PS4 ) continue;
            kcs    = csp_ics[klcs];
            lcs    = csp_jcs[klcs];
            klps0  = csp_leading_ps_pair[klcs];
            nklps  = csp_leading_ps_pair[klcs+1]-klps0;
            kat    = shel_atm[kcs];
            lat    = shel_atm[lcs];
            kao0   = shel_ini[kcs];
            lao0   = shel_ini[lcs];
            C[0]=atom_x[kat]; C[1]=atom_y[kat]; C[2]=atom_z[kat];
            D[0]=atom_x[lat]; D[1]=atom_y[lat]; D[2]=atom_z[lat];
            for ( i=0; i<3; i++ ) {
                AC[i] = A[i] - C[i];
                DC[i] = D[i] - C[i];
            }
            gpu_twoint_core_os_dsps(
                    &La, &Lb, &Lc, &Ld,
                    &nijps, &psp_zeta[ijps0], &psp_dkps[ijps0],
                    &psp_xiza[ijps0], BA,
                    &nklps, &psp_zeta[klps0], &psp_dkps[klps0],
                    &psp_xiza[klps0], DC,   AC,      DINTEG );
            ipat=((Lab != Lcd)||(ics==kcs && jcs>lcs) ? true : false );
            for ( i=0, iao=iao0, ix=0; i<6; i++, iao++ ) {
                I2 = (iao*iao+iao)>>1;
                for ( j=0, jao=jao0; j<1; j++, jao++ ) {
                    if ( jao>iao ) { ix+=3*1; continue; }
                    IJ = I2 + jao;
                    coe0 = ( iao==jao ? HALF : ONE );
                    for ( k=0, kao=kao0; k<3; k++, kao++ ) {
                        K2 = (kao*kao+kao)>>1;
                        for ( l=0, lao=lao0; l<1; l++, lao++, ix++ ) {
                            if ( lao>kao ) continue;
                            if ( fabs(DINTEG[ix]) > EPS_ERI ) {
                                KL = K2 + lao;
                                if ( IJ >= KL ) {
                                    coe = coe0;
                                    if ( kao==lao ) coe *= HALF;
                                    if ( KL == IJ ) coe *= HALF;
                                    ebuf_val[nzeri]     = coe*DINTEG[ix];
                                    ebuf_ind4[nzeri4+0] = (short int)iao;
                                    ebuf_ind4[nzeri4+1] = (short int)jao;
                                    ebuf_ind4[nzeri4+2] = (short int)kao;
                                    ebuf_ind4[nzeri4+3] = (short int)lao;
                                    nzeri++;
                                    nzeri4+=4;
                                } else if ( ipat ) {
                                    coe = coe0;
                                    if ( kao==lao ) coe*=HALF;
                                    ebuf_val[nzeri]     = coe*DINTEG[ix];
                                    ebuf_ind4[nzeri4+0] = (short int)kao;
                                    ebuf_ind4[nzeri4+1] = (short int)lao;
                                    ebuf_ind4[nzeri4+2] = (short int)iao;
                                    ebuf_ind4[nzeri4+3] = (short int)jao;
                                    nzeri++;
                                    nzeri4+=4;
                                }
                            }
                        }	// l
                    }	// k
                }	// j
            }	// i
            if ( nzeri >= max_nzeri ) {
                *last_ijcs = ijcs;
                *last_klcs = klcs;
                *ebuf_non_zero_eri = nzeri;
                return OFMO_EBUF_FULL;
            }
        }	// for (klcs)
    }	// for (ijcs)
    *ebuf_non_zero_eri = nzeri;
    return OFMO_EBUF_NOFULL;
}

int gpu_twoint_direct_os_dsps(
        const int *pnworkers, const int *pworkerid,
        const int *pLa, const int *pLb, const int *pLc, const int *pLd,
        const int shel_atm[], const int shel_ini[],
        const double atom_x[], const double atom_y[],
        const double atom_z[], const int leading_cs_pair[],
        const double csp_schwarz[],
        const int csp_ics[], const int csp_jcs[],
        const int csp_leading_ps_pair[],
        const double psp_zeta[], const double psp_dkps[],
        const double psp_xiza[],
        // for direct SCF
        const long *petmp_max_nzeri, long *petmp_non_zero_eri,
        double etmp_val[], short int etmp_ind4[],
        const int *plast_ijcs, const int *plast_klcs,
        // density matrix & G-matrix data
        const int *pnao, const double Ds[], double G[] ) {
    int nworkers=*pnworkers, workerid=*pworkerid;
    int La=*pLa, Lb=*pLb, Lc=*pLc, Ld=*pLd;
    int last_ijcs=*plast_ijcs, last_klcs=*plast_klcs, nao=*pnao;
    long max_nzeri=*petmp_max_nzeri;
    long nzeri4, nzeri=*petmp_non_zero_eri;
    //
    int Lab, Lcd, i, j, k, l, ipat, ix;
    int I2, IJ, K2, KL;
    int ijcs, ijcs0, ijcs1;
    int klcs, klcs0, klcs1, max_klcs;
    int ijps0, nijps, klps0, nklps;
    int ics, iat, iao, iao0, jcs, jat, jao, jao0;
    int kcs, kat, kao, kao0, lcs, lat, lao, lao0;
    double A[3], B[3], C[3], D[3], BA[3], DC[3], AC[3];
    double val_ab, val_cd, coe, coe0;
    double DINTEG[6*1*3*1];
    int mythread;

    mythread = omp_get_thread_num();
    if ( DFACT == NULL ) DFACT = gpu_getadd_dfact();
    
    Lab = La*(La+1)/2+Lb;
    Lcd = Lc*(Lc+1)/2+Ld;
    ijcs1 = leading_cs_pair[Lab+1];
    klcs0 = leading_cs_pair[Lcd];
    klcs1 = leading_cs_pair[Lcd+1];
    if ( last_ijcs != -1 ) { 
        ijcs = last_ijcs;
        klcs = last_klcs+1;
    } else {
        ijcs = leading_cs_pair[Lab] + workerid;
        klcs = klcs0;
    }
    
    max_nzeri -= 6*1*3*1;
    nzeri4    = nzeri*4;
    if ( nzeri >= max_nzeri ) {
        gpu_integ_add_fock( nao, nzeri, etmp_val, etmp_ind4, Ds, G );
        nzeri = nzeri4 = 0;
    }
    
    for ( ; ijcs<ijcs1; ijcs+=nworkers ) {
        val_ab = csp_schwarz[ijcs];
        ics    = csp_ics[ijcs];
        jcs    = csp_jcs[ijcs];
        ijps0  = csp_leading_ps_pair[ijcs];
        nijps  = csp_leading_ps_pair[ijcs+1]-ijps0;
        iat    = shel_atm[ics];
        jat    = shel_atm[jcs];
        iao0   = shel_ini[ics];
        jao0   = shel_ini[jcs];
        A[0]=atom_x[iat]; A[1]=atom_y[iat]; A[2]=atom_z[iat];
        B[0]=atom_x[jat]; B[1]=atom_y[jat]; B[2]=atom_z[jat];
        for ( i=0; i<3; i++ ) BA[i] = B[i] - A[i];
        max_klcs = ( Lab == Lcd ? ijcs+1 : klcs1 );
        for ( ; klcs<max_klcs; klcs++ ) {
            val_cd = csp_schwarz[klcs];
            if ( val_ab*val_cd < EPS_PS4 ) continue;
            kcs    = csp_ics[klcs];
            lcs    = csp_jcs[klcs];
            klps0  = csp_leading_ps_pair[klcs];
            nklps  = csp_leading_ps_pair[klcs+1]-klps0;
            kat    = shel_atm[kcs];
            lat    = shel_atm[lcs];
            kao0   = shel_ini[kcs];
            lao0   = shel_ini[lcs];
            C[0]=atom_x[kat]; C[1]=atom_y[kat]; C[2]=atom_z[kat];
            D[0]=atom_x[lat]; D[1]=atom_y[lat]; D[2]=atom_z[lat];
            for ( i=0; i<3; i++ ) {
                AC[i] = A[i] - C[i];
                DC[i] = D[i] - C[i];
            }
            gpu_twoint_core_os_dsps(
                    &La, &Lb, &Lc, &Ld,
                    &nijps, &psp_zeta[ijps0], &psp_dkps[ijps0],
                    &psp_xiza[ijps0], BA,
                    &nklps, &psp_zeta[klps0], &psp_dkps[klps0],
                    &psp_xiza[klps0], DC,   AC,      DINTEG );
            ipat=((Lab != Lcd)||(ics==kcs && jcs>lcs) ? true : false );
            for ( i=0, iao=iao0, ix=0; i<6; i++, iao++ ) {
                I2 = (iao*iao+iao)>>1;
                for ( j=0, jao=jao0; j<1; j++, jao++ ) {
                    if ( jao>iao ) { ix+=3*1; continue; }
                    IJ = I2 + jao;
                    coe0 = ( iao==jao ? HALF : ONE );
                    for ( k=0, kao=kao0; k<3; k++, kao++ ) {
                        K2 = (kao*kao+kao)>>1;
                        for ( l=0, lao=lao0; l<1; l++, lao++, ix++ ) {
                            if ( lao>kao ) continue;
                            if ( fabs(DINTEG[ix]) > EPS_ERI ) {
                                KL = K2 + lao;
                                if ( IJ >= KL ) {
                                    coe = coe0;
                                    if ( kao==lao ) coe *= HALF;
                                    if ( KL == IJ ) coe *= HALF;
                                    etmp_val[nzeri]     = coe*DINTEG[ix];
                                    etmp_ind4[nzeri4+0] = (short int)iao;
                                    etmp_ind4[nzeri4+1] = (short int)jao;
                                    etmp_ind4[nzeri4+2] = (short int)kao;
                                    etmp_ind4[nzeri4+3] = (short int)lao;
                                    nzeri++;
                                    nzeri4+=4;
                                } else if ( ipat ) {
                                    coe = coe0;
                                    if ( kao==lao ) coe*=HALF;
                                    etmp_val[nzeri]     = coe*DINTEG[ix];
                                    etmp_ind4[nzeri4+0] = (short int)kao;
                                    etmp_ind4[nzeri4+1] = (short int)lao;
                                    etmp_ind4[nzeri4+2] = (short int)iao;
                                    etmp_ind4[nzeri4+3] = (short int)jao;
                                    nzeri++;
                                    nzeri4+=4;
                                }
                            }
                        }	// l
                    }	// k
                }	// j
            }	// i
            if ( nzeri >= max_nzeri ) {
                gpu_integ_add_fock( nao, nzeri, etmp_val, etmp_ind4,
                        Ds, G );
                nzeri = nzeri4= 0;
            }
        }	// for (klcs)
        klcs = klcs0;
    }	// for (ijcs)
    *petmp_non_zero_eri = nzeri;
    return 0;
}
#endif // 0
